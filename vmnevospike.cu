#include "hip/hip_runtime.h"


/*
*  evospikegen.cu
*  
*  Created by Tom Clayton and Duncan MacGregor.
*  University of Edinburgh 2016
*  Released under MIT license, see https://opensource.org/licenses/MIT
*
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <mersenne.h>


#define BLOCK_DIM 16


__global__ void spikenetGPU(
	float *d_fChrome,
	int	d_iChromeCount,
	float d_fRunTime, 
	float d_fCutOffTop,
	float d_fInitBinWidth,
	float d_fBinIncAmount,
	float *d_oInts,
	float *d_oISI,
	float *d_oTemp,
	float *d_oInputVars,
	float *d_oMeanIntraBurstNoise,
	float *d_oMeanExtraBurstNoise,
	int *d_iThreadData) 
{
	/////////////////////////////////////////////////////////////////////////////////////////////////////
	// access thread id + No thread stuff.  This is used for addressing
	const unsigned int tid = threadIdx.x;
	const unsigned int num_threads = blockDim.x;
	const unsigned int bid = blockIdx.x;


	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Copy Control Parameters (CP) that are in perpetual use to local memory

	// Synaptic Input (IPSPs and EPSPs)
	
	float P_ESynRate				=	d_fChrome[0*d_iChromeCount + (tid + bid*num_threads)];
	float P_ISynRate				=	d_fChrome[1*d_iChromeCount + (tid + bid*num_threads)];
	float P_PSPMag					=	d_fChrome[2*d_iChromeCount + (tid + bid*num_threads)];
	float P_PSPDecay				=	d_fChrome[3*d_iChromeCount + (tid + bid*num_threads)];
	float P_RestToThreshold			=	d_fChrome[4*d_iChromeCount + (tid + bid*num_threads)];

	// Post Spike Potentials
	float P_kHAP					=	d_fChrome[5*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauHAP					=	d_fChrome[6*d_iChromeCount + (tid + bid*num_threads)];
	float P_kDAP					=	d_fChrome[7*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauDAP					=	d_fChrome[8*d_iChromeCount + (tid + bid*num_threads)];
	float P_kAHP					=	d_fChrome[9*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauAHP					=	d_fChrome[10*d_iChromeCount + (tid + bid*num_threads)];


	d_iThreadData[0 + 4*tid + bid*num_threads*4] = (int)P_ESynRate;
	d_iThreadData[1 + 4*tid + bid*num_threads*4] = tid;
	d_iThreadData[2 + 4*tid + bid*num_threads*4] = num_threads;
	d_iThreadData[3 + 4*tid + bid*num_threads*4] = bid;

	
	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Pre Calculate the stuff that is used to find the correct bin.  Most of this is just the parts of
	//the quadratic solver equation (-b + sqrt(b*b -4ac))/2a.
	float b = d_fInitBinWidth - d_fBinIncAmount/2;
	float BSquared = b*b;
	float FourA = 2*d_fBinIncAmount;
	float TwoA = d_fBinIncAmount;

	//Create look-up table for poisson Distribution
	float InputE[8], InputI[8];
	float ExpectedNoE = P_ESynRate / 1000.0f;
	float ExpectedNoI = P_ISynRate / 1000.0f;
	float Factorial = 1;
	float CumulativeE = 0;
	float CumulativeI = 0;
	for(float i=0; i<8; i++)
	{
		Factorial = 1;
		for(int j=1; j<i+1; j++)
		{
			Factorial *= j;
		}

		CumulativeE += (pow(ExpectedNoE, i) * pow(2.718281828f, -ExpectedNoE)) / Factorial;
		InputE[(int)i] = CumulativeE;

		CumulativeI += (pow(ExpectedNoI, i) * pow(2.718281828f, -ExpectedNoI)) / Factorial;
		InputI[(int)i] = CumulativeI;
	}

	unsigned int RandExcit = 234248430 - tid + bid*num_threads;
	unsigned int RandInhib = 936753243 - tid + bid*num_threads;


	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Create the Variables (HAP, DAP, AHP, LastDAP, Drift, Membrane Potential, Inhibition, Balance Point)

	//Input
	float V_Input = 0;

	//Post Spike Potentials
	float V_HAP	= 0;
	float V_DAP = 0;
	float V_AHP = 0;

	//Spikes
	float Spike	= 0;
	float LastSpike	= 0;

	//Max average of 500 spikes per second
	int MaxSpikes = ((int)(d_fRunTime/1000)) * 500;

	__syncthreads();

	int Spikes = 0;

	//clear ISI memory locations
	for(int i = 0; i < 512; i++)
		d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;

	//Input
	d_oInputVars[0 + tid*4 + bid*num_threads*4] = 0.0f; 
	d_oInputVars[1 + tid*4 + bid*num_threads*4] = 0.0f; 
	d_oInputVars[2 + tid*4 + bid*num_threads*4] = 0.0f; 
	d_oInputVars[3 + tid*4 + bid*num_threads*4] = 0.0f; 

	__syncthreads();

	////////////////////////////////////////////////////////////
	//Model Starts Here
	for(float step=0; step < d_fRunTime; step++)
	{	
		RandExcit = (22695477 * RandExcit + 1) & 0xFFFFFFFF;
		RandInhib = (22695477 * RandInhib + 1) & 0xFFFFFFFF;

		float fRandExcit = RandExcit / 4294967295.0f;
		float fRandInhib = RandInhib / 4294967295.0f;

		float Total = 0;
		for(int i=0; i<8; i++)
		{
			if (fRandExcit > InputE[i]) Total += P_PSPMag;
			if (fRandInhib > InputI[i]) Total -= P_PSPMag;
		}

		V_Input = V_Input - V_Input * P_PSPDecay + Total;


		//Calculate the Exponential Decays; Decay to the membrane rest

		V_HAP = V_HAP - V_HAP * P_tauHAP + P_kHAP * Spike;
		V_DAP = V_DAP - V_DAP * P_tauDAP + P_kDAP * Spike;
		V_AHP = V_AHP - V_AHP * P_tauAHP + P_kAHP * Spike;


		//Check to see if it has fired
		if((V_Input - V_HAP + V_DAP - V_AHP) > P_RestToThreshold) {
			Spike = 1;

			//If the model fires record and timestamp the event
			//float Diff = (step - LastSpike) * 0.001f;
			float Diff = step - LastSpike;

			if(Spikes < 16384) d_oInts[Spikes + tid*16384 + bid*num_threads*16384] = step;           
			
			if((Diff > 2) && (Diff < d_fCutOffTop)) {
				//This equation defines an ever increasing bin width
				//Even though it is quantised, it will increase by BinSpacing
				//every bin.
				float fBinNo = (-b + sqrt(BSquared + FourA * Diff)) / TwoA;
				int iBinNo = fBinNo;
				//Increment the relevent bin within Memory
				if((iBinNo < 512) & (iBinNo > 2)) d_oISI[iBinNo + tid*512 + bid*num_threads*512]++;
			}
			LastSpike = step;
			Spikes++;
		}
		else Spike = 0;

		if(Spikes > MaxSpikes) break;
	}

	////////////////////   End of model loop

	__syncthreads();


	// Spike time analysis - generate quad binned ISI histogram

	//reset the temp
	for(int i=0; i<512; i++) {
		d_oTemp[i + tid*512 + bid*num_threads*512] = 0.0f;
	} 

	//Smooth and Scale ISI Histogram
	float NumBinnedEvents = 0;
	for(int i=0; i<512; i++) {
		float Div = 0.0f;

		for (int j = (i-2); j < (i+3); j++) {	
			if((j > -1)&(j < 512)) {
				d_oTemp[i + tid*512 + bid*num_threads*512] += d_oISI[j + tid*512 + bid*num_threads*512];
				Div++;
			}
		}
		Div = d_oTemp[i + tid*512 + bid*num_threads*512] / Div;
		NumBinnedEvents += Div;
		d_oTemp[i + tid*512 + bid*num_threads*512] = Div;
	}

	__syncthreads();

	if(NumBinnedEvents > 0.0f) {
		for(int i = 0; i < 512; i++)
			d_oISI[i + tid*512 + bid*num_threads*512] = d_oTemp[i + tid*512 + bid*num_threads*512]/NumBinnedEvents;
	}
	else {
		for(int i = 0; i < 512; i++) d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;
	}

	//Save the Noise Value;
	d_oMeanIntraBurstNoise[tid + bid*num_threads] = d_oInputVars[0 + tid*4 + bid*num_threads*4]/d_oInputVars[1 + tid*4 + bid*num_threads*4];
	d_oMeanExtraBurstNoise[tid + bid*num_threads] = d_oInputVars[2 + tid*4 + bid*num_threads*4]/d_oInputVars[3 + tid*4 + bid*num_threads*4];

	__syncthreads();

	//If there are too many events... clean the Exp
	if(Spikes > MaxSpikes) {
		for(int i=0; i<512; i++)
			d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;
	}


	d_oMeanExtraBurstNoise[tid + bid*num_threads] = Spikes;
	//d_oMeanExtraBurstNoise[tid + bid*num_threads] = P_SynRate;
	__syncthreads();	
}


__global__ void spikevmnnetGPU(
	int numcells,
	float *d_fChrome,
	int d_iChromeCount,
	int d_iRunTime,
	float d_fCutOffTop,
	float d_fInitBinWidth,
	float d_fBinIncAmount,
	float *d_oInts,
	float *d_oISI,
	float *d_oTemp,
	float *d_oSpikeCounts,
	hiprandState *state)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int num_threads = blockDim.x;
	const unsigned int bid = blockIdx.x;

	// Read neuron chrome
	float P_ESynRate				=	d_fChrome[0*d_iChromeCount + (tid + bid*num_threads)];
	float P_ISynRate				=	d_fChrome[1*d_iChromeCount + (tid + bid*num_threads)];
	float P_PSPMag					=	d_fChrome[2*d_iChromeCount + (tid + bid*num_threads)];
	float P_PSPDecay				=	d_fChrome[3*d_iChromeCount + (tid + bid*num_threads)];
	float P_RestToThreshold			=	d_fChrome[4*d_iChromeCount + (tid + bid*num_threads)];

	// Post Spike Potentials
	float P_kHAP					=	d_fChrome[5*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauHAP					=	d_fChrome[6*d_iChromeCount + (tid + bid*num_threads)];
	float P_kDAP					=	d_fChrome[7*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauDAP					=	d_fChrome[8*d_iChromeCount + (tid + bid*num_threads)];
	float P_kAHP					=	d_fChrome[9*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauAHP					=	d_fChrome[10*d_iChromeCount + (tid + bid*num_threads)];

	// Network parameters
	float esynL1					=   d_fChrome[11*d_iChromeCount + (tid + bid*num_threads)];
	float esynweight				=   d_fChrome[12*d_iChromeCount + (tid + bid*num_threads)];
	float syndelay					=   d_fChrome[13*d_iChromeCount + (tid + bid*num_threads)];
	float syndelrange			    =   d_fChrome[14*d_iChromeCount + (tid + bid*num_threads)];



	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Pre Calculate the stuff that is used to find the correct bin.  Most of this is just the parts of
	//the quadratic solver equation (-b + sqrt(b*b -4ac))/2a.
	float b = d_fInitBinWidth - d_fBinIncAmount/2;
	float BSquared = b*b;
	float FourA = 2*d_fBinIncAmount;
	float TwoA = d_fBinIncAmount;


	float nepsp, nipsp;
	float ExpectedNoE = P_ESynRate / 1000.0f;
	float ExpectedNoI = P_ISynRate / 1000.0f;

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState randstate = state[id];



	// Tom Random Poisson Code
	//Create look-up table for poisson Distribution
	float InputE[8], InputI[8];
	float Factorial = 1;
	float CumulativeE = 0;
	float CumulativeI = 0;
	for(float i=0; i<8; i++)
	{
		Factorial = 1;
		for(int j=1; j<i+1; j++)
		{
			Factorial *= j;
		}

		CumulativeE += (pow(ExpectedNoE, i) * pow(2.718281828f, -ExpectedNoE)) / Factorial;
		InputE[(int)i] = CumulativeE;

		CumulativeI += (pow(ExpectedNoI, i) * pow(2.718281828f, -ExpectedNoI)) / Factorial;
		InputI[(int)i] = CumulativeI;
	}

	unsigned int RandExcit = 234248430 - tid + bid*num_threads;
	unsigned int RandInhib = 936753243 - tid + bid*num_threads;

	int i;
	//numcells = 1;

	// Network Data
	//float esynL1 = 0.6;
	float esyntrans = 0.5;
	//float syndelay = 5;
	//float syndelrange = 0;
	//float esynweight = 1;

	unsigned char econnect[50];
	unsigned char enetwork[50][50];
	unsigned char esynqueue[50][20];
	unsigned char activity[50];
	
	float esynsum[50];
	int Spikes[50];


	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Create the Variables (HAP, DAP, AHP, LastDAP, Drift, Membrane Potential, Inhibition, Balance Point)


	//Input
	float V_Input[50];

	//Post Spike Potentials
	float V_HAP[50];
	float V_AHP[50];
	float V_DAP[50];

	// Initialise neuron variables
	for(i=0; i<numcells; i++) { 
		V_Input[i] = 0;
		V_HAP[i] = 0;
		V_DAP[i] = 0;
		V_AHP[i] = 0;
		activity[i] = 0;
		Spikes[i] = 0;
	}

	//Spikes
	//float Spike	= 0;
	float LastSpike	= 0;

	//Max average of 500 spikes per second
	int MaxSpikes = ((int)(d_iRunTime/1000)) * 500;

	__syncthreads();


	//clear ISI memory locations
	for(i = 0; i < 512; i++) d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;

	__syncthreads();


	float epspt = 0;
    float ipspt = 0;

	// Network Generation
	for(i=0; i<numcells; i++) {
		econnect[i] = 0;
		for(float j=0; j<numcells; j++) {
			float d = hiprand_uniform(&randstate);
			if(d <= esynL1 && i != j) enetwork[i][econnect[i]++] = j;
		}
		for(int j=0; j<20; j++) esynqueue[i][j] = 0;       // queue max length fixed at 20
	}


	////////////////////////////////////////////////////////////
	// Model Loop
	for(float step=0; step < d_iRunTime; step++) {	
	
		// Network Input
		for(i=0; i<numcells; i++) {
			// Add network activity to input queue
			for(int c=0; c<econnect[i]; c++) 
				if(activity[enetwork[i][c]] == 1) {
					float synrand = hiprand_uniform(&randstate);
					if(esyntrans >= synrand) { 
						float syndel = (syndelay - 1) + (syndelrange + 1) * (synrand * (1/esyntrans));
						esynqueue[i][(int)syndel] = esynqueue[i][(int)syndel] + esynweight;	
					}
				}

			//if(activity[i]) activity[i] = activity[i] - 1;

			// Read and shift input queue
			esynsum[i] = esynqueue[i][0];
			for(int j=0; j<20-1; j++) esynqueue[i][j] = esynqueue[i][j+1];
			esynqueue[i][20-1] = 0;
		}
		

		// Membrane Activity and Spiking
		for(i=0; i<numcells; i++) {

			// Tom Poisson Code
			RandExcit = (22695477 * RandExcit + 1) & 0xFFFFFFFF;
			RandInhib = (22695477 * RandInhib + 1) & 0xFFFFFFFF;

			float fRandExcit = RandExcit / 4294967295.0f;
			float fRandInhib = RandInhib / 4294967295.0f;

			float Total = 0;
			for(int p=0; p<8; p++)
			{
				if (fRandExcit > InputE[p]) Total += P_PSPMag;
				if (fRandInhib > InputI[p]) Total -= P_PSPMag;
			}
		

			// CUDA Poisson Generator

			//nepsp = hiprand_poisson(&randstate, (double)ExpectedNoE);
			//nipsp = hiprand_poisson(&randstate, (double)ExpectedNoI);
		

			// CUDA Uniform Generator
			/*
			nepsp = 0;
			if(ExpectedNoE > 0) {
				while(epspt < 1) {
					nepsp++;
					epspt = -log(1 - hiprand_uniform(&randstate)) / ExpectedNoE + epspt;
				}
				epspt = epspt - 1;
			}

			nipsp = 0;
			if(ExpectedNoI > 0) {
				while(ipspt < 1) {
					nipsp++;
					ipspt = -log(1 - hiprand_uniform(&randstate)) / ExpectedNoI + ipspt;
				}
				ipspt = ipspt - 1;
			}
		

			float Total = P_PSPMag * nepsp - P_PSPMag * nipsp;*/

			float NetInput = P_PSPMag * esynsum[i];
			//float NetInput = 0;
		
			V_Input[i] = V_Input[i] - V_Input[i] * P_PSPDecay + Total + NetInput;


			//Calculate the Exponential Decays; Decay to the membrane rest

			V_HAP[i] = V_HAP[i] - V_HAP[i] * P_tauHAP + P_kHAP * activity[i];
			V_DAP[i] = V_DAP[i] - V_DAP[i] * P_tauDAP + P_kDAP * activity[i];
			V_AHP[i] = V_AHP[i] - V_AHP[i] * P_tauAHP + P_kAHP * activity[i];


			//Check to see if it has fired
			if((V_Input[i] - V_HAP[i] + V_DAP[i] - V_AHP[i]) > P_RestToThreshold) {
				//Spike = 1;
				activity[i] = 1;

				if(i == 0) {
					//If the model fires record and timestamp the event
					//float Diff = (step - LastSpike) * 0.001f;
					float Diff = step - LastSpike;

					if(Spikes[i] < 16384) d_oInts[Spikes[i] + tid*16384 + bid*num_threads*16384] = step;           
			
					if((Diff > 2) && (Diff < d_fCutOffTop)) {
						//This equation defines an ever increasing bin width
						//Even though it is quantised, it will increase by BinSpacing
						//every bin.
						float fBinNo = (-b + sqrt(BSquared + FourA * Diff)) / TwoA;
						int iBinNo = fBinNo;
						//Increment the relevent bin within Memory
						if((iBinNo < 512) & (iBinNo > 2)) d_oISI[iBinNo + tid*512 + bid*num_threads*512]++;
					}
					LastSpike = step;	
				}
				Spikes[i]++;

			}
			else {
				//Spike = 0;
				activity[i] = 0;
			}		
		}
		if(Spikes[0] > MaxSpikes) break;
	}

	////////////////////   End of model loop

	__syncthreads();


	// Spike time analysis - generate quad binned ISI histogram

	//reset the temp
	for(int i=0; i<512; i++) {
		d_oTemp[i + tid*512 + bid*num_threads*512] = 0.0f;
	} 

	//Smooth and Scale ISI Histogram
	float NumBinnedEvents = 0;
	for(int i=0; i<512; i++) {
		float Div = 0.0f;

		for (int j = (i-2); j < (i+3); j++) {	
			if((j > -1)&(j < 512)) {
				d_oTemp[i + tid*512 + bid*num_threads*512] += d_oISI[j + tid*512 + bid*num_threads*512];
				Div++;
			}
		}
		Div = d_oTemp[i + tid*512 + bid*num_threads*512] / Div;
		NumBinnedEvents += Div;
		d_oTemp[i + tid*512 + bid*num_threads*512] = Div;
	}

	__syncthreads();

	if(NumBinnedEvents > 0.0f) {
		for(int i = 0; i < 512; i++)
			d_oISI[i + tid*512 + bid*num_threads*512] = d_oTemp[i + tid*512 + bid*num_threads*512]/NumBinnedEvents;
	}
	else {
		for(int i = 0; i < 512; i++) d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;
	}

	
	__syncthreads();

	//If there are too many events... clean the Exp
	if(Spikes[0] > MaxSpikes) {
		for(int i=0; i<512; i++)
			d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;
	}


	for(i=0; i<numcells; i++) d_oSpikeCounts[i + tid*128 + bid*num_threads*128] = Spikes[i];
	
	__syncthreads();	



	/*
	// hiprand test code

	hiprandState_t state;

	hiprand_init(0, 0, 0, &state);

	for(int i=0; i<10; i++) d_oInts[i + tid*16384 + bid*num_threads*16384] = hiprand(&state)*(1.0/UINT_MAX);
	d_oSpikeCounts[tid + bid*num_threads] = 10;

	__syncthreads();	
	*/
}


__global__ void spikegenvmnGPU(
	float *d_fChrome,
	int d_iChromeCount,
	int d_iRunTime,
	float d_fCutOffTop,
	float d_fInitBinWidth,
	float d_fBinIncAmount,
	float *d_oInts,
	float *d_oISI,
	float *d_oTemp,
	float *d_oSpikeCounts,
	hiprandState *state)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int num_threads = blockDim.x;
	const unsigned int bid = blockIdx.x;

	// Read neuron chrome
	float P_ESynRate				=	d_fChrome[0*d_iChromeCount + (tid + bid*num_threads)];
	float P_ISynRate				=	d_fChrome[1*d_iChromeCount + (tid + bid*num_threads)];
	float P_PSPMag					=	d_fChrome[2*d_iChromeCount + (tid + bid*num_threads)];
	float P_PSPDecay				=	d_fChrome[3*d_iChromeCount + (tid + bid*num_threads)];
	float P_RestToThreshold			=	d_fChrome[4*d_iChromeCount + (tid + bid*num_threads)];

	// Post Spike Potentials
	float P_kHAP					=	d_fChrome[5*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauHAP					=	d_fChrome[6*d_iChromeCount + (tid + bid*num_threads)];
	float P_kDAP					=	d_fChrome[7*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauDAP					=	d_fChrome[8*d_iChromeCount + (tid + bid*num_threads)];
	float P_kAHP					=	d_fChrome[9*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauAHP					=	d_fChrome[10*d_iChromeCount + (tid + bid*num_threads)];


	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Pre Calculate the stuff that is used to find the correct bin.  Most of this is just the parts of
	//the quadratic solver equation (-b + sqrt(b*b -4ac))/2a.
	float b = d_fInitBinWidth - d_fBinIncAmount/2;
	float BSquared = b*b;
	float FourA = 2*d_fBinIncAmount;
	float TwoA = d_fBinIncAmount;

	float nepsp, nipsp;
	float ExpectedNoE = P_ESynRate / 1000.0f;
	float ExpectedNoI = P_ISynRate / 1000.0f;

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState randstate = state[id];


	// Tom Random Poisson Code
	//Create look-up table for poisson Distribution
	float InputE[8], InputI[8];
	float Factorial = 1;
	float CumulativeE = 0;
	float CumulativeI = 0;
	for(float i=0; i<8; i++)
	{
		Factorial = 1;
		for(int j=1; j<i+1; j++)
		{
			Factorial *= j;
		}

		CumulativeE += (pow(ExpectedNoE, i) * pow(2.718281828f, -ExpectedNoE)) / Factorial;
		InputE[(int)i] = CumulativeE;

		CumulativeI += (pow(ExpectedNoI, i) * pow(2.718281828f, -ExpectedNoI)) / Factorial;
		InputI[(int)i] = CumulativeI;
	}

	unsigned int RandExcit = 234248430 - tid + bid*num_threads;
	unsigned int RandInhib = 936753243 - tid + bid*num_threads;

	
	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Create the Variables (HAP, DAP, AHP, LastDAP, Drift, Membrane Potential, Inhibition, Balance Point)


	//Input
	float V_Input = 0;

	//Post Spike Potentials
	float V_HAP = 0;
	float V_AHP = 0;
	float V_DAP = 0;

	//Spikes
	float Spike	= 0;
	float LastSpike	= 0;

	//Max average of 500 spikes per second
	int MaxSpikes = ((int)(d_iRunTime/1000)) * 500;

	__syncthreads();

	int Spikes = 0;

	//clear ISI memory locations
	for(int i=0; i<512; i++) d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;

	__syncthreads();


	////////////////////////////////////////////////////////////
	//Model Starts Here
	for(float step=0; step < d_iRunTime; step++) {

		// Membrane Activity and Spiking
		
		// Tom Poisson Code
		RandExcit = (22695477 * RandExcit + 1) & 0xFFFFFFFF;
		RandInhib = (22695477 * RandInhib + 1) & 0xFFFFFFFF;

		float fRandExcit = RandExcit / 4294967295.0f;
		float fRandInhib = RandInhib / 4294967295.0f;

		float Total = 0;
		for(int p=0; p<8; p++) {
			if (fRandExcit > InputE[p]) Total += P_PSPMag;
			if (fRandInhib > InputI[p]) Total -= P_PSPMag;
		}
		

		// CUDA Poisson Generator

		//nepsp = hiprand_poisson(&randstate, (double)ExpectedNoE);
		//nipsp = hiprand_poisson(&randstate, (double)ExpectedNoI);
		

		// CUDA Uniform Generator
		/*
		nepsp = 0;
		if(ExpectedNoE > 0) {
			while(epspt < 1) {
				nepsp++;
				epspt = -log(1 - hiprand_uniform(&randstate)) / ExpectedNoE + epspt;
			}
			epspt = epspt - 1;
		}

		nipsp = 0;
		if(ExpectedNoI > 0) {
			while(ipspt < 1) {
				nipsp++;
				ipspt = -log(1 - hiprand_uniform(&randstate)) / ExpectedNoI + ipspt;
			}
			ipspt = ipspt - 1;
		}
		

		float Total = P_PSPMag * nepsp - P_PSPMag * nipsp;*/

		//float NetInput = P_PSPMag * esynsum[i];
		float NetInput = 0;
		
		V_Input = V_Input - V_Input * P_PSPDecay + Total;


		//Calculate the Exponential Decays; Decay to the membrane rest

		V_HAP = V_HAP - V_HAP * P_tauHAP + P_kHAP * Spike;
		V_DAP = V_DAP - V_DAP * P_tauDAP + P_kDAP * Spike;
		V_AHP = V_AHP - V_AHP * P_tauAHP + P_kAHP * Spike;


		//Check to see if it has fired
		if((V_Input - V_HAP + V_DAP - V_AHP) > P_RestToThreshold) {
			Spike = 1;
			
			//If the model fires record and timestamp the event
			//float Diff = (step - LastSpike) * 0.001f;
			float Diff = step - LastSpike;

			if(Spikes < 16384) d_oInts[Spikes + tid*16384 + bid*num_threads*16384] = step;           
			
			if((Diff > 2) && (Diff < d_fCutOffTop)) {
				//This equation defines an ever increasing bin width
				//Even though it is quantised, it will increase by BinSpacing
				//every bin.
				float fBinNo = (-b + sqrt(BSquared + FourA * Diff)) / TwoA;
				int iBinNo = fBinNo;
				//Increment the relevent bin within Memory
				if((iBinNo < 512) & (iBinNo > 2)) d_oISI[iBinNo + tid*512 + bid*num_threads*512]++;
			}
			LastSpike = step;
			Spikes++;
		}
		else Spike = 0;

	    if(Spikes > MaxSpikes) break;
	}

	////////////////////   End of model loop

	__syncthreads();


	// Spike time analysis - generate quad binned ISI histogram

	//reset the temp
	for(int i=0; i<512; i++) {
		d_oTemp[i + tid*512 + bid*num_threads*512] = 0.0f;
	} 

	//Smooth and Scale ISI Histogram
	float NumBinnedEvents = 0;
	for(int i=0; i<512; i++) {
		float Div = 0.0f;

		for (int j = (i-2); j < (i+3); j++) {	
			if((j > -1)&(j < 512)) {
				d_oTemp[i + tid*512 + bid*num_threads*512] += d_oISI[j + tid*512 + bid*num_threads*512];
				Div++;
			}
		}
		Div = d_oTemp[i + tid*512 + bid*num_threads*512] / Div;
		NumBinnedEvents += Div;
		d_oTemp[i + tid*512 + bid*num_threads*512] = Div;
	}

	__syncthreads();

	if(NumBinnedEvents > 0.0f) {
		for(int i = 0; i < 512; i++)
			d_oISI[i + tid*512 + bid*num_threads*512] = d_oTemp[i + tid*512 + bid*num_threads*512]/NumBinnedEvents;
	}
	else {
		for(int i = 0; i < 512; i++) d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;
	}

	
	__syncthreads();

	//If there are too many events... clean the Exp
	if(Spikes > MaxSpikes) {
		for(int i=0; i<512; i++)
			d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;
	}


	d_oSpikeCounts[tid + bid*num_threads] = Spikes;
	
	__syncthreads();	



	/*
	// hiprand test code

	hiprandState_t state;

	hiprand_init(0, 0, 0, &state);

	for(int i=0; i<10; i++) d_oInts[i + tid*16384 + bid*num_threads*16384] = hiprand(&state)*(1.0/UINT_MAX);
	d_oSpikeCounts[tid + bid*num_threads] = 10;

	__syncthreads();	
	*/
}


__global__ void randinitGPU(unsigned long seed, hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}


void spikevmnnetCPU(
	int numcells,
	float *d_fChrome,
	int d_iChromeCount,
	int d_iRunTime,
	float d_fCutOffTop,
	float d_fInitBinWidth,
	float d_fBinIncAmount,
	float *d_oInts,
	float *d_oISI,
	float *d_oTemp,
	float *d_oSpikeCounts)
{
	//const unsigned int tid = threadIdx.x;
	//const unsigned int num_threads = blockDim.x;
	//const unsigned int bid = blockIdx.x;

	const unsigned int tid = 0;
	const unsigned int num_threads = 1;
	const unsigned int bid = 0;

	int i, j;

	FILE *ofp;
	ofp = fopen("netcpu.txt", "w");
	fprintf(ofp, "netcpu diagnostics\n\n");

	// Read neuron chrome
	float P_ESynRate				=	d_fChrome[0*d_iChromeCount + (tid + bid*num_threads)];
	float P_ISynRate				=	d_fChrome[1*d_iChromeCount + (tid + bid*num_threads)];
	float P_PSPMag					=	d_fChrome[2*d_iChromeCount + (tid + bid*num_threads)];
	float P_PSPDecay				=	d_fChrome[3*d_iChromeCount + (tid + bid*num_threads)];
	float P_RestToThreshold			=	d_fChrome[4*d_iChromeCount + (tid + bid*num_threads)];

	// Post Spike Potentials
	float P_kHAP					=	d_fChrome[5*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauHAP					=	d_fChrome[6*d_iChromeCount + (tid + bid*num_threads)];
	float P_kDAP					=	d_fChrome[7*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauDAP					=	d_fChrome[8*d_iChromeCount + (tid + bid*num_threads)];
	float P_kAHP					=	d_fChrome[9*d_iChromeCount + (tid + bid*num_threads)];
	float P_tauAHP					=	d_fChrome[10*d_iChromeCount + (tid + bid*num_threads)];

	// Network parameters
	float esynL1					=   d_fChrome[11*d_iChromeCount + (tid + bid*num_threads)];
	float esynweight				=   d_fChrome[12*d_iChromeCount + (tid + bid*num_threads)];
	float syndelay					=   d_fChrome[13*d_iChromeCount + (tid + bid*num_threads)];
	float syndelrange			    =   d_fChrome[14*d_iChromeCount + (tid + bid*num_threads)];



	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Pre Calculate the stuff that is used to find the correct bin.  Most of this is just the parts of
	//the quadratic solver equation (-b + sqrt(b*b -4ac))/2a.
	float b = d_fInitBinWidth - d_fBinIncAmount/2;
	float BSquared = b*b;
	float FourA = 2*d_fBinIncAmount;
	float TwoA = d_fBinIncAmount;


	float nepsp, nipsp;
	float ExpectedNoE = P_ESynRate / 1000.0f;
	float ExpectedNoI = P_ISynRate / 1000.0f;

	int id = tid + bid * num_threads;
	//hiprandState randstate = state[id];



	// Tom Random Poisson Code
	//Create look-up table for poisson Distribution
	float InputE[8], InputI[8];
	float Factorial = 1;
	float CumulativeE = 0;
	float CumulativeI = 0;
	for(i=0; i<8; i++)
	{
		Factorial = 1;
		for(j=1; j<i+1; j++)
		{
			Factorial *= j;
		}

		CumulativeE += (pow(ExpectedNoE, i) * pow(2.718281828f, -ExpectedNoE)) / Factorial;
		InputE[(int)i] = CumulativeE;

		CumulativeI += (pow(ExpectedNoI, i) * pow(2.718281828f, -ExpectedNoI)) / Factorial;
		InputI[(int)i] = CumulativeI;
	}

	unsigned int RandExcit = 234248430 - tid + bid*num_threads;
	unsigned int RandInhib = 936753243 - tid + bid*num_threads;

	//numcells = 1;

	// Network Data
	//float esynL1 = 0.6;
	float esyntrans = 0.5;
	//float syndelay = 5;
	//float syndelrange = 0;
	//float esynweight = 1;

	unsigned char econnect[50];
	unsigned char enetwork[50][50];
	unsigned char esynqueue[50][20];
	unsigned char activity[50];
	
	float esynsum[50];
	int Spikes[50];


	/////////////////////////////////////////////////////////////////////////////////////////////////////
	//Create the Variables (HAP, DAP, AHP, LastDAP, Drift, Membrane Potential, Inhibition, Balance Point)


	//Input
	float V_Input[50];

	//Post Spike Potentials
	float V_HAP[50];
	float V_AHP[50];
	float V_DAP[50];

	// Initialise neuron variables
	for(i=0; i<numcells; i++) { 
		V_Input[i] = 0;
		V_HAP[i] = 0;
		V_DAP[i] = 0;
		V_AHP[i] = 0;
		activity[i] = 0;
		Spikes[i] = 0;
	}

	//Spikes
	//float Spike	= 0;
	float LastSpike	= 0;

	//Max average of 500 spikes per second
	int MaxSpikes = ((int)(d_iRunTime/1000)) * 500;

	//__syncthreads();


	//clear ISI memory locations
	for(i = 0; i < 512; i++) d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;

	//__syncthreads();


	float epspt = 0;
    float ipspt = 0;

	// Network Generation
	for(i=0; i<numcells; i++) {
		econnect[i] = 0;
		for(j=0; j<numcells; j++) {
			float d = mrand01(); // hiprand_uniform(&randstate);
			if(d <= esynL1 && i != j) enetwork[i][econnect[i]++] = j;
		}
		for(j=0; j<20; j++) esynqueue[i][j] = 0;       // queue max length fixed at 20
	}


	////////////////////////////////////////////////////////////
	// Model Loop
	for(float step=0; step < d_iRunTime; step++) {	
	
		// Network Input
		for(i=0; i<numcells; i++) {
			// Add network activity to input queue
			for(int c=0; c<econnect[i]; c++) 
				if(activity[enetwork[i][c]] == 1) {
					float synrand = mrand01(); //hiprand_uniform(&randstate);
					if(esyntrans >= synrand) { 
						float syndel = (syndelay - 1) + (syndelrange + 1) * (synrand * (1/esyntrans));
						esynqueue[i][(int)syndel] = esynqueue[i][(int)syndel] + esynweight;	
					}
				}

			//if(activity[i]) activity[i] = activity[i] - 1;

			// Read and shift input queue
			esynsum[i] = esynqueue[i][0];
			for(j=0; j<20-1; j++) esynqueue[i][j] = esynqueue[i][j+1];
			esynqueue[i][20-1] = 0;
		}
		

		// Membrane Activity and Spiking
		for(i=0; i<numcells; i++) {

			// Tom Poisson Code
			RandExcit = (22695477 * RandExcit + 1) & 0xFFFFFFFF;
			RandInhib = (22695477 * RandInhib + 1) & 0xFFFFFFFF;

			float fRandExcit = RandExcit / 4294967295.0f;
			float fRandInhib = RandInhib / 4294967295.0f;

			float Total = 0;
			for(int p=0; p<8; p++)
			{
				if (fRandExcit > InputE[p]) Total += P_PSPMag;
				if (fRandInhib > InputI[p]) Total -= P_PSPMag;
			}
		

			// CUDA Poisson Generator

			//nepsp = hiprand_poisson(&randstate, (double)ExpectedNoE);
			//nipsp = hiprand_poisson(&randstate, (double)ExpectedNoI);
		

			// CUDA Uniform Generator
			/*
			nepsp = 0;
			if(ExpectedNoE > 0) {
				while(epspt < 1) {
					nepsp++;
					epspt = -log(1 - hiprand_uniform(&randstate)) / ExpectedNoE + epspt;
				}
				epspt = epspt - 1;
			}

			nipsp = 0;
			if(ExpectedNoI > 0) {
				while(ipspt < 1) {
					nipsp++;
					ipspt = -log(1 - hiprand_uniform(&randstate)) / ExpectedNoI + ipspt;
				}
				ipspt = ipspt - 1;
			}
		

			float Total = P_PSPMag * nepsp - P_PSPMag * nipsp;*/

			float NetInput = P_PSPMag * esynsum[i];
			//float NetInput = 0;
		
			V_Input[i] = V_Input[i] - V_Input[i] * P_PSPDecay + Total + NetInput;


			//Calculate the Exponential Decays; Decay to the membrane rest

			V_HAP[i] = V_HAP[i] - V_HAP[i] * P_tauHAP + P_kHAP * activity[i];
			V_DAP[i] = V_DAP[i] - V_DAP[i] * P_tauDAP + P_kDAP * activity[i];
			V_AHP[i] = V_AHP[i] - V_AHP[i] * P_tauAHP + P_kAHP * activity[i];


			//Check to see if it has fired
			if((V_Input[i] - V_HAP[i] + V_DAP[i] - V_AHP[i]) > P_RestToThreshold) {
				//Spike = 1;
				activity[i] = 1;

				if(i == 0) {
					//If the model fires record and timestamp the event
					//float Diff = (step - LastSpike) * 0.001f;
					float Diff = step - LastSpike;

					if(Spikes[i] < 16384) d_oInts[Spikes[i] + tid*16384 + bid*num_threads*16384] = step;           
			
					if((Diff > 2) && (Diff < d_fCutOffTop)) {
						//This equation defines an ever increasing bin width
						//Even though it is quantised, it will increase by BinSpacing
						//every bin.
						float fBinNo = (-b + sqrt(BSquared + FourA * Diff)) / TwoA;
						int iBinNo = fBinNo;
						//Increment the relevent bin within Memory
						if((iBinNo < 512) & (iBinNo > 2)) d_oISI[iBinNo + tid*512 + bid*num_threads*512]++;
					}
					LastSpike = step;	
				}
				Spikes[i]++;

			}
			else {
				//Spike = 0;
				activity[i] = 0;
			}		
		}
		if(Spikes[0] > MaxSpikes) break;
	}

	////////////////////   End of model loop

	//__syncthreads();


	// Spike time analysis - generate quad binned ISI histogram

	//reset the temp
	for(i=0; i<512; i++) {
		d_oTemp[i + tid*512 + bid*num_threads*512] = 0.0f;
	} 

	//Smooth and Scale ISI Histogram
	float NumBinnedEvents = 0;
	for(i=0; i<512; i++) {
		float Div = 0.0f;

		for(j=(i-2); j<(i+3); j++) {	
			if((j > -1)&(j < 512)) {
				d_oTemp[i + tid*512 + bid*num_threads*512] += d_oISI[j + tid*512 + bid*num_threads*512];
				Div++;
			}
		}
		Div = d_oTemp[i + tid*512 + bid*num_threads*512] / Div;
		NumBinnedEvents += Div;
		d_oTemp[i + tid*512 + bid*num_threads*512] = Div;
	}

	//__syncthreads();

	if(NumBinnedEvents > 0.0f) {
		for(i = 0; i < 512; i++)
			d_oISI[i + tid*512 + bid*num_threads*512] = d_oTemp[i + tid*512 + bid*num_threads*512]/NumBinnedEvents;
	}
	else {
		for(i = 0; i < 512; i++) d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;
	}

	
	//__syncthreads();

	//If there are too many events... clean the Exp
	if(Spikes[0] > MaxSpikes) {
		for(i=0; i<512; i++)
			d_oISI[i + tid*512 + bid*num_threads*512] = 0.0f;
	}


	for(i=0; i<numcells; i++) d_oSpikeCounts[i + tid*128 + bid*num_threads*128] = Spikes[i];
	
	//__syncthreads();	
	
	//fprintf(ofp, "\nesyn %d  isyn %d\n", esyncount/1000, isyncount/1000);
	fprintf(ofp, "\nSpikes %d\n", Spikes[0]);
	//fprintf(ofp, "\nSpikes %.0f\n", d_oMeanExtraBurstNoise[tid + bid*num_threads]);
	for(i=0; i<10; i++) fprintf(ofp, "Spike %d Index %d Time %.2f\n", i, i + tid*16384 + bid*num_threads*16384, d_oInts[i + tid*16384 + bid*num_threads*16384]);
	fclose(ofp);
}


void EvoFitVMN_GPU(int runmode, int numcells, float *chromepop, int paramcount, int threadcount, int blocksize, float runtime, float *Ints, float *ISIs, float *SpikeCounts)
{
	float *d_chromepop;
	float *d_Ints, *d_SpikeCounts;
	float *d_ISIs, *d_Temp;
	unsigned long spikeseed;

	hiprandState *state;

    hipSetDevice(0);

	hipMallocManaged((void **) &d_chromepop, threadcount * paramcount * sizeof(float));
	hipMallocManaged((void **) &d_Ints, threadcount * 512 * 32 * sizeof(float));
	hipMallocManaged((void **) &d_ISIs, threadcount * 512 * sizeof(float));
	hipMallocManaged((void **) &d_Temp, threadcount * 512 * sizeof(float));
	hipMallocManaged((void **) &d_SpikeCounts, threadcount * 128 * sizeof(float));
	hipMemcpy(d_chromepop, chromepop, threadcount * paramcount * sizeof(float), hipMemcpyHostToDevice);
	
	int blockSize = blocksize;
	int blocks = threadcount / blockSize;

	dim3 grid(1, 1, 1);
	dim3 threads(threadcount, 1, 1);

	float *Temp = new float[threadcount * 512];

	FILE *ofp;
	ofp = fopen("netgpu.txt", "w");
	fprintf(ofp, "netgpu diagnostics\n\n");

	// Allocate space for Poisson RNG states on device
    hipMallocManaged((void **) &state, threadcount * sizeof(hiprandState));
	// Initialise PRNGs
	spikeseed = (unsigned long)time(NULL);
	randinitGPU<<< blocks, blocksize>>>(spikeseed, state);

	if(runmode == 0) {
		fprintf(ofp, "running CPU net mode\n");
		spikevmnnetCPU(numcells, chromepop, threadcount, runtime, 7052, 1, 0.05, Ints, ISIs, Temp, SpikeCounts);
	}
	else if(runmode == 1) spikegenvmnGPU<<< blocks, blockSize >>>(d_chromepop, threadcount, runtime, 7052, 1, 0.05, d_Ints, d_ISIs, d_Temp, d_SpikeCounts, state);
	else {
		fprintf(ofp, "running gpu net mode\n");
		spikevmnnetGPU<<< blocks, blockSize >>>(numcells, d_chromepop, threadcount, runtime, 7052, 1, 0.05, d_Ints, d_ISIs, d_Temp, d_SpikeCounts, state);
	}
	
	if(runmode) {
		hipMemcpy(Ints, d_Ints, threadcount * 512 * 32 * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(ISIs, d_ISIs, threadcount * 512 * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(SpikeCounts, d_SpikeCounts, threadcount * 128 * sizeof(float), hipMemcpyDeviceToHost);
	}
	
	hipFree(d_chromepop);
	hipFree(d_Ints);
	hipFree(d_ISIs);
	hipFree(d_Temp);
	hipFree(d_SpikeCounts);

	delete[] Temp;

	fclose(ofp);
}